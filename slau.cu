#include "hip/hip_runtime.h"
#include "slau.h"

__constant__ int const_n;

__device__ void print_matrix(double *a, int n)
{
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			printf("%f ", a[n * i + j]);
		}
		printf("\n");
	}
}

__device__ void print_matrix(int *a, int n)
{
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			printf("%d ", a[n * i + j]);
		}
		printf("\n");
	}
}

__device__ void transform_matrix(double *a, int n)
{
	double divider;

	for (int k = 0; k < n - 1; k++)
	{
		for (int i = k; i < n - 1; i++)
		{
			divider = a[n * (i + 1) + k] / a[n * k + k];
			for (int j = 0; j < n; j++)
			{
				a[n * (i + 1) + k + j] -= divider * a[n * k + k + j];
			}
		}
	}
}

__device__ int get_det(double *a, int n)
{
	double det;

	if (n == 0)
		n = N;
	else
		n = const_n;
	transform_matrix(a, n);
	print_matrix(a, n);
	det = 1;
	for (int i = 0; i < n; i++)
	{
		det *= a[n * i + i];
	}
	printf("det = %d\n", (int) round(det));
	return ((int) round(det));
}

__device__ void init_sub_a(double *a, double *sub_a, int r, int c)
{
	int idx_sub_a;

	idx_sub_a = 0;
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			if (i == r || j == c)
				continue;
			sub_a[idx_sub_a++] = a[N * i + j];
		}
	}
}

__global__ void search_det(double *a, int *det)
{
	*det = get_det(a, 0);
	// printf("const_n = %d\n", const_n);
}

__global__ void search_minor_matrix(double *a, double *sub_a, int *minor)
{
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			init_sub_a(a, sub_a, i, j);
			minor[N * i + j] = get_det(sub_a, 1);
		}
	}
	printf("Матрица миноров\n");
	print_matrix(minor, N);
}

int	main()
{
	double  host_a[SIZE] = {2, 3, 4, 1};
	// double  host_a[SIZE] = {1, -2, 3, 4, 90, 6, -7, 8, 9};
	// double host_a[SIZE] = {5, 3, 21, 7, 4, 47, 12, 18, 77, 45, 3, 1, -6, 90, 34, -82};
	// double host_a[SIZE] = {5, 3, 21, 7, 4, 47, 12, 18, 77, 45, 3, 1, -6, 90, 34, -82, -103, 71, 51, 21, 33, -367, 16, 2, 1};
	// int host_b[N] = {8, 6};
	int *host_minor;
	int host_det;
	double *dev_a;
	double *dev_sub_a;
	int *dev_det;
	int *dev_minor;
	int host_n;
	int	int_size;
	int double_size;

	int_size = sizeof(int);
	double_size = sizeof(double);
	host_minor = (int *) malloc(int_size * SIZE);
	host_n = get_n();
	hipMalloc(&dev_a, double_size * SIZE);
	hipMalloc(&dev_sub_a, double_size * host_n);
	hipMalloc(&dev_det, int_size);
	hipMalloc(&dev_minor, int_size * SIZE);

	hipMemcpyToSymbol(HIP_SYMBOL(const_n), &host_n, int_size, 0, hipMemcpyHostToDevice);
	hipMemcpy(dev_a, host_a, double_size * SIZE, hipMemcpyHostToDevice);

	search_det<<<1, 1>>>(dev_a, dev_det);
	hipMemcpy(&host_det, dev_det, int_size, hipMemcpyDeviceToHost);
	printf("Определитель матрицы А = %d\n", host_det);
	hipMemcpy(dev_a, host_a, double_size * SIZE, hipMemcpyHostToDevice);
	search_minor_matrix<<<1, 1>>>(dev_a, dev_sub_a, dev_minor);

	free(host_minor);
	hipFree(dev_a);
	hipFree(dev_sub_a);
	hipFree(dev_det);
	hipFree(dev_minor);

	check_cuda_error("");
	return 0;
}
