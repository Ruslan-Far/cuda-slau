
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#define N 2
#define SIZE N * N

__device__ void print_matrix(double *a)
{
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			printf("%f ", a[N * i + j]);
		}
		printf("\n");
	}
}

__device__ void transform_matrix(double *a)
{
	double divider;

	for (int k = 0; k < N - 1; k++)
	{
		for (int i = k; i < N - 1; i++)
		{
			divider = a[N * (i + 1) + k] / a[N * k + k];
			for (int j = 0; j < N; j++)
			{
				a[N * (i + 1) + k + j] -= divider * a[N * k + k + j];
			}
		}
	}
}

__device__ int get_det(double *a)
{
	double det;

	transform_matrix(a);
	print_matrix(a);
	det = 1;
	for (int i = 0; i < N; i++)
	{
		det *= a[N * i + i];
	}
	return ((int) round(det));
}

__global__ void search_det_and_minor_matrix(double *a, int *det)
{
	*det = get_det(a);
}

void check_cuda_error(const char *msg)
{
    hipError_t err = hipGetLastError();

    if (hipSuccess != err)
	{
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
    }
}

int	main()
{
	double  host_a[SIZE] = {2, 3, 4, 1};
	// double  host_a[SIZE] = {1, -2, 3, 4, 90, 6, -7, 8, 9};
	// double host_a[SIZE] = {5, 3, 21, 7, 4, 47, 12, 18, 77, 45, 3, 1, -6, 90, 34, -82};
	// double host_a[SIZE] = {5, 3, 21, 7, 4, 47, 12, 18, 77, 45, 3, 1, -6, 90, 34, -82, -103, 71, 51, 21, 33, -367, 16, 2, 1};
	// int host_b[N] = {8, 6};
	int host_det;
	double *dev_a;
	int *dev_det;
	int	int_size;
	int double_size;

	int_size = sizeof(int);
	double_size = sizeof(double);
	hipMalloc(&dev_a, double_size * SIZE);
	hipMalloc(&dev_det, int_size);
	hipMemcpy(dev_a, host_a, double_size * SIZE, hipMemcpyHostToDevice);
	search_det_and_minor_matrix<<<1, 1>>>(dev_a, dev_det);
	hipMemcpy(&host_det, dev_det, int_size, hipMemcpyDeviceToHost);
	printf("Определитель матрицы А = %d\n", host_det);
	hipFree(dev_a);
	hipFree(dev_det);

	check_cuda_error("");
	return 0;
}
