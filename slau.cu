#include "hip/hip_runtime.h"
#include "slau.h"

__constant__ int const_n;

__device__ int def_n(int n)
{
	if (n == 0)
		return N;
	return const_n;
}

__global__ void search_det(double *a, int *det)
{
	*det = get_det(a, 0);
}

__global__ void search_minor_algaddit_matrix(double *a, double *sub_a, int *minor_algaddit)
{
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			init_sub_a(a, sub_a, i, j);
			minor_algaddit[N * i + j] = get_det(sub_a, 1);
		}
	}
	printf("Матрица миноров\n");
	print_matrix(minor_algaddit);
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			minor_algaddit[N * i + j] *= pow(-1, i + j);
		}
	}
}

__global__ void transpose_matrix(int *a, double *at)
{
	int a_idx = N * (blockDim.y * blockIdx.y + threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;
	if (a_idx >= SIZE)
	{
		printf("a_idx = %d\n", a_idx);
		return;
	}
	int at_idx = N * (blockDim.x * blockIdx.x + threadIdx.x) + blockDim.y * blockIdx.y + threadIdx.y;
	if (at_idx >= SIZE)
	{
		printf("at_idx = %d\n", at_idx);
		return;
	}
	at[at_idx] = a[a_idx];
}

void init_dim3(dim3 *blocksPerGrid, dim3 *threadsPerBlock)
{
	if (N <= BLOCK_N)
	{
		*blocksPerGrid = dim3(1);
		*threadsPerBlock = dim3(N, N);
	}
	else
	{
		if (N % BLOCK_N == 0)
			*blocksPerGrid = dim3(N / BLOCK_N, N / BLOCK_N);
		else
			*blocksPerGrid = dim3(N / BLOCK_N + 1, N / BLOCK_N + 1);
		*threadsPerBlock = dim3(BLOCK_N, BLOCK_N);
	}
}

int	main()
{
	double  host_a[SIZE] = {2, 3, 4, 1};
	// double  host_a[SIZE] = {1, -2, 3, 4, 90, 6, -7, 8, 9};
	// double host_a[SIZE] = {5, 3, 21, 7, 4, 47, 12, 18, 77, 45, 3, 1, -6, 90, 34, -82};
	// double host_a[SIZE] = {5, 3, 21, 7, 4, 47, 12, 18, 77, 45, 3, 1, -6, 90, 34, -82, -103, 71, 51, 21, 33, -367, 16, 2, 1};
	// int host_b[N] = {8, 6};
	int *host_minor_algaddit;
	int host_det;
	double *dev_a;
	double *dev_sub_a;
	int *dev_det;
	int *dev_minor_algaddit;
	int host_n;
	int	int_size;
	int double_size;
	dim3 blocksPerGrid;
	dim3 threadsPerBlock;

	int_size = sizeof(int);
	double_size = sizeof(double);
	host_n = get_n();
	host_minor_algaddit = (int *) malloc(int_size * SIZE);
	// blocksPerGrid = dim3(N / BLOCK_N + 1, N / BLOCK_N + 1);
	// threadsPerBlock = dim3(BLOCK_N, BLOCK_N);
	init_dim3(&blocksPerGrid, &threadsPerBlock);

	hipMalloc(&dev_a, double_size * SIZE);
	hipMalloc(&dev_sub_a, double_size * host_n);
	hipMalloc(&dev_det, int_size);
	hipMalloc(&dev_minor_algaddit, int_size * SIZE);

	hipMemcpyToSymbol(HIP_SYMBOL(const_n), &host_n, int_size, 0, hipMemcpyHostToDevice);
	hipMemcpy(dev_a, host_a, double_size * SIZE, hipMemcpyHostToDevice);

	search_det<<<1, 1>>>(dev_a, dev_det);
	hipMemcpy(&host_det, dev_det, int_size, hipMemcpyDeviceToHost);
	printf("Определитель матрицы = %d\n", host_det);
	hipMemcpy(dev_a, host_a, double_size * SIZE, hipMemcpyHostToDevice);
	search_minor_algaddit_matrix<<<1, 1>>>(dev_a, dev_sub_a, dev_minor_algaddit);
	hipMemcpy(host_minor_algaddit, dev_minor_algaddit, int_size * SIZE, hipMemcpyDeviceToHost);
	printf("Матрица алгебраических дополнений\n");
	host_print_matrix(host_minor_algaddit);
	transpose_matrix<<<blocksPerGrid, threadsPerBlock>>>(dev_minor_algaddit, dev_a);
	hipMemcpy(host_a, dev_a, double_size * SIZE, hipMemcpyDeviceToHost);
	printf("Транспонированная матрица\n");
	host_print_matrix(host_a);

	free(host_minor_algaddit);
	hipFree(dev_a);
	hipFree(dev_sub_a);
	hipFree(dev_det);
	hipFree(dev_minor_algaddit);

	check_cuda_error("");

	return 0;
}
