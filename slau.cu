
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#define N 4
#define SIZE N * N

__device__ float fact(int n)
{
	int factorial = 1;

	while (n > 1)
	{
		factorial *= n;
		n -= 1;
	}
	return factorial;
}

__global__ void searchDet(int *a, int *det)
{
	int term;
	int n_fact;

	n_fact = fact(N);
	printf("n_fact = %d\n", n_fact);
	*det = 0;
	for (int k = 0; k < n_fact; k++)
	{
		term = pow(-1, k);
		if (term > 0)
		{
			for (int i = 0, cpk = k; i < N; i++)
			{
				printf("%d\n", a[N * i + cpk % N]);
				term *= a[N * i + cpk % N];
				cpk++;
			}
		}
		else
		{
			for (int i = 0, cpk = k; i < N; i++)
			{
				printf("%d\n", a[N * i + cpk % N]);
				term *= a[N * i + cpk % N];
				cpk--;
				if (cpk < 0)
					cpk = N - 1;
			}
		}
		printf("term = %d\n\n\n", term);
		*det += term;
	}
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();

    if (hipSuccess != err)
	{
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
    }
}

int	main()
{
	// int host_a[SIZE] = {2, 3, 4, 1};
	// int host_a[SIZE] = {1, -2, 3, 4, 90, 6, -7, 8, 9};
	int host_a[SIZE] = {5, 3, 21, 7, 4, 47, 12, 18, 77, 45, 3, 1, -6, 90, 34, -82};
	// int host_b[N] = {8, 6};
	int host_det;
	int *dev_a;
	int *dev_det;
	int	size;

	size = sizeof(int);
	host_det = 0;
	hipMalloc(&dev_a, size * SIZE);
	hipMalloc(&dev_det, size);
	hipMemcpy(dev_a, host_a, size * SIZE, hipMemcpyHostToDevice);
	searchDet<<<1, 1>>>(dev_a, dev_det);
	hipMemcpy(&host_det, dev_det, size, hipMemcpyDeviceToHost);
	printf("Определитель матрицы А = %d\n", host_det);
	hipFree(dev_a);
	hipFree(dev_det);

	checkCUDAError("");
	return 0;
}
