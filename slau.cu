
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#define N 2
#define SIZE N * N

__device__ float fact(int n)
{
	int factorial = 1;

	while (n > 1)
	{
		factorial *= n;
		n -= 1;
	}
	return factorial;
}

__device__ void print_matrix(double *a)
{
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			printf("%f ", a[N * i + j]);
		}
		printf("\n");
	}
}

__device__ void transform_matrix(double *a)
{
	double divider;

	for (int k = 0; k < N - 1; k++)
	{
		for (int i = k; i < N - 1; i++)
		{
			divider = a[N * (i + 1) + k] / a[N * k + k];
			// printf("divider = %f\n", divider);
			for (int j = 0; j < N; j++)
			{
				a[N * (i + 1) + k + j] -= divider * a[N * k + k + j];
			}
		}
	}
}

__global__ void search_det(double *a, double *det)
{
	transform_matrix(a);
	print_matrix(a);
	*det = 1;
	for (int i = 0; i < N; i++)
	{
		*det *= a[N * i + i];
	}
	printf("%f\n", *det);
	// int term;
	// int n_fact;

	// n_fact = fact(N);
	// printf("n_fact = %d\n", n_fact);
	// *det = 0;
	// for (int k = 0; k < n_fact; k++)
	// {
	// 	term = pow(-1, k);
	// 	if (term > 0)
	// 	{
	// 		for (int i = 0, cpk = k; i < N; i++)
	// 		{
	// 			printf("%d\n", a[N * i + cpk % N]);
	// 			term *= a[N * i + cpk % N];
	// 			cpk++;
	// 		}
	// 	}
	// 	else
	// 	{
	// 		for (int i = 0, cpk = k; i < N; i++)
	// 		{
	// 			printf("%d\n", a[N * i + cpk % N]);
	// 			term *= a[N * i + cpk % N];
	// 			cpk--;
	// 			if (cpk < 0)
	// 				cpk = N - 1;
	// 		}
	// 	}
	// 	printf("term = %d\n\n\n", term);
	// 	*det += term;
	// }
}

void check_cuda_error(const char *msg)
{
    hipError_t err = hipGetLastError();

    if (hipSuccess != err)
	{
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
    }
}

int	main()
{
	double  host_a[SIZE] = {2, 3, 4, 1};
	// double  host_a[SIZE] = {1, -2, 3, 4, 90, 6, -7, 8, 9};
	// double host_a[SIZE] = {5, 3, 21, 7, 4, 47, 12, 18, 77, 45, 3, 1, -6, 90, 34, -82};
	// double host_a[SIZE] = {5, 3, 21, 7, 4, 47, 12, 18, 77, 45, 3, 1, -6, 90, 34, -82, -103, 71, 51, 21, 33, -367, 16, 2, 1};
	// int host_b[N] = {8, 6};
	double host_det;
	double *dev_a;
	double *dev_det;
	int	int_size;
	int double_size;

	int_size = sizeof(int);
	double_size = sizeof(double);
	hipMalloc(&dev_a, double_size * SIZE);
	hipMalloc(&dev_det, double_size);
	hipMemcpy(dev_a, host_a, double_size * SIZE, hipMemcpyHostToDevice);
	search_det<<<1, 1>>>(dev_a, dev_det);
	hipMemcpy(&host_det, dev_det, double_size, hipMemcpyDeviceToHost);
	host_det = round(host_det);
	printf("Определитель матрицы А = %f\n", host_det);
	hipFree(dev_a);
	hipFree(dev_det);

	check_cuda_error("");
	return 0;
}
