
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#define N 2
#define SIZE N * N

__device__ void print_matrix(double *a, int n)
{
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			printf("%f ", a[n * i + j]);
		}
		printf("\n");
	}
}

__device__ void transform_matrix(double *a, int n)
{
	double divider;

	for (int k = 0; k < n - 1; k++)
	{
		for (int i = k; i < n - 1; i++)
		{
			divider = a[n * (i + 1) + k] / a[n * k + k];
			for (int j = 0; j < n; j++)
			{
				a[n * (i + 1) + k + j] -= divider * a[n * k + k + j];
			}
		}
	}
}

__device__ int get_det(double *a, int n)
{
	double det;

	if (n == 0)
		n = N;
	transform_matrix(a, n);
	print_matrix(a, n);
	det = 1;
	for (int i = 0; i < n; i++)
	{
		det *= a[n * i + i];
	}
	return ((int) round(det));
}

__device__ int get_n()
{
	int k;

	k = 1;
	for (int i = 1; i < SIZE; i += k)
	{
		k += 2;
	}
	printf("__fsqrt_rn(SIZE - k) = %d\n", (int) __fsqrt_rn(SIZE - k));
	return __fsqrt_rn(SIZE - k);
}

__device__ void init_sub_a(double *a, double *sub_a, int i, int j)
{

}

__global__ void search_det(double *a, int *det)
{
	*det = get_det(a, 0);
}

__global__ void search_minor_matrix(double *a, int *minor)
{
	double *sub_a;
	int n;

	n = get_n();
	hipMalloc(&sub_a, sizeof(double) * SIZE);
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			init_sub_a(a, sub_a, i, j);
			minor[N * i + j] = get_det(sub_a, n);
		}
	}
	hipFree(sub_a);
}

void check_cuda_error(const char *msg)
{
    hipError_t err = hipGetLastError();

    if (hipSuccess != err)
	{
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
    }
}

int	main()
{
	double  host_a[SIZE] = {2, 3, 4, 1};
	// double  host_a[SIZE] = {1, -2, 3, 4, 90, 6, -7, 8, 9};
	// double host_a[SIZE] = {5, 3, 21, 7, 4, 47, 12, 18, 77, 45, 3, 1, -6, 90, 34, -82};
	// double host_a[SIZE] = {5, 3, 21, 7, 4, 47, 12, 18, 77, 45, 3, 1, -6, 90, 34, -82, -103, 71, 51, 21, 33, -367, 16, 2, 1};
	// int host_b[N] = {8, 6};
	int *host_minor;
	int host_det;
	double *dev_a;
	double *dev_sub_a;
	int *dev_det;
	int *dev_minor;
	int	int_size;
	int double_size;

	int_size = sizeof(int);
	double_size = sizeof(double);
	host_minor = (int *) malloc(int_size * SIZE);
	hipMalloc(&dev_a, double_size * SIZE);
	hipMalloc(&dev_sub_a, double_size * SIZE);
	hipMalloc(&dev_det, int_size);
	hipMalloc(&dev_minor, int_size * SIZE);

	hipMemcpy(dev_a, host_a, double_size * SIZE, hipMemcpyHostToDevice);
	search_det<<<1, 1>>>(dev_a, dev_det);
	hipMemcpy(&host_det, dev_det, int_size, hipMemcpyDeviceToHost);
	printf("Определитель матрицы А = %d\n", host_det);
	// cudaMemcpy(dev_a, host_a, double_size * SIZE, cudaMemcpyHostToDevice);
	// search_minor_matrix<<<1, 1>>>(dev_a, dev_minor);

	free(host_minor);
	hipFree(dev_a);
	hipFree(dev_det);

	check_cuda_error("");
	return 0;
}
