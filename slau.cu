#include "hip/hip_runtime.h"
#include "slau.h"

__constant__ int const_n;

__device__ int def_n(int n)
{
	if (n == 0)
		return N;
	return const_n;
}

__global__ void search_det(double *a, int *det)
{
	*det = get_det(a, 0);
}

__global__ void search_minor_algaddit_matrix(double *a, double *sub_a, int *minor_algaddit)
{
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			init_sub_a(a, sub_a, i, j);
			minor_algaddit[N * i + j] = get_det(sub_a, 1);
		}
	}
	printf("Матрица миноров\n");
	print_matrix(minor_algaddit);
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			minor_algaddit[N * i + j] *= pow(-1, i + j);
		}
	}
}

int	main()
{
	double  host_a[SIZE] = {2, 3, 4, 1};
	// double  host_a[SIZE] = {1, -2, 3, 4, 90, 6, -7, 8, 9};
	// double host_a[SIZE] = {5, 3, 21, 7, 4, 47, 12, 18, 77, 45, 3, 1, -6, 90, 34, -82};
	// double host_a[SIZE] = {5, 3, 21, 7, 4, 47, 12, 18, 77, 45, 3, 1, -6, 90, 34, -82, -103, 71, 51, 21, 33, -367, 16, 2, 1};
	// int host_b[N] = {8, 6};
	int *host_minor_algaddit;
	int host_det;
	double *dev_a;
	double *dev_sub_a;
	int *dev_det;
	int *dev_minor_algaddit;
	int host_n;
	int	int_size;
	int double_size;

	int_size = sizeof(int);
	double_size = sizeof(double);
	host_n = get_n();
	host_minor_algaddit = (int *) malloc(int_size * SIZE);
	hipMalloc(&dev_a, double_size * SIZE);
	hipMalloc(&dev_sub_a, double_size * host_n);
	hipMalloc(&dev_det, int_size);
	hipMalloc(&dev_minor_algaddit, int_size * SIZE);

	hipMemcpyToSymbol(HIP_SYMBOL(const_n), &host_n, int_size, 0, hipMemcpyHostToDevice);
	hipMemcpy(dev_a, host_a, double_size * SIZE, hipMemcpyHostToDevice);

	search_det<<<1, 1>>>(dev_a, dev_det);
	hipMemcpy(&host_det, dev_det, int_size, hipMemcpyDeviceToHost);
	printf("Определитель матрицы А = %d\n", host_det);
	hipMemcpy(dev_a, host_a, double_size * SIZE, hipMemcpyHostToDevice);
	search_minor_algaddit_matrix<<<1, 1>>>(dev_a, dev_sub_a, dev_minor_algaddit);
	hipMemcpy(host_minor_algaddit, dev_minor_algaddit, int_size * SIZE, hipMemcpyDeviceToHost);
	printf("Матрица алгебраических дополнений\n");
	host_print_matrix(host_minor_algaddit);

	free(host_minor_algaddit);
	hipFree(dev_a);
	hipFree(dev_sub_a);
	hipFree(dev_det);
	hipFree(dev_minor_algaddit);

	check_cuda_error("");
	return 0;
}
