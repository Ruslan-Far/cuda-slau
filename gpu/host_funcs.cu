#include "hip/hip_runtime.h"
#include "slau.h"

void	host_check_cuda_error(const char *msg)
{
    hipError_t err = hipGetLastError();

    if (hipSuccess != err)
	{
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
    }
}

void	host_init_dim3(dim3 *blocksPerGrid, dim3 *threadsPerBlock)
{
	if (N <= BLOCK_N)
	{
		*blocksPerGrid = dim3(1);
		*threadsPerBlock = dim3(N, N);
	}
	else
	{
		if (N % BLOCK_N == 0)
			*blocksPerGrid = dim3(N / BLOCK_N, N / BLOCK_N);
		else
			*blocksPerGrid = dim3(N / BLOCK_N + 1, N / BLOCK_N + 1);
		*threadsPerBlock = dim3(BLOCK_N, BLOCK_N);
	}
}

void	host_init_a(double *a)
{
	for (int i = 0; i < SIZE; i++)
	{
		a[i] = rand();
		if (a[i] > 100000)
			a[i] = round(a[i] / 100000000);
	}
}

void	host_init_b(int *b)
{
	for (int i = 0; i < N; i++)
		b[i] = N - i;
}
