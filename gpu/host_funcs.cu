#include "hip/hip_runtime.h"
#include "slau.h"

void host_check_cuda_error(const char *msg)
{
    hipError_t err = hipGetLastError();

    if (hipSuccess != err)
	{
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
    }
}

int host_get_n()
{
	int k;

	k = 1;
	for (int i = 1; i < SIZE; i += k)
	{
		k += 2;
	}
	return sqrt(SIZE - k);
}

void host_init_dim3(dim3 *blocksPerGrid, dim3 *threadsPerBlock)
{
	if (N <= BLOCK_N)
	{
		*blocksPerGrid = dim3(1);
		*threadsPerBlock = dim3(N, N);
	}
	else
	{
		if (N % BLOCK_N == 0)
			*blocksPerGrid = dim3(N / BLOCK_N, N / BLOCK_N);
		else
			*blocksPerGrid = dim3(N / BLOCK_N + 1, N / BLOCK_N + 1);
		*threadsPerBlock = dim3(BLOCK_N, BLOCK_N);
	}
}

void host_init_a(double *a)
{
	for (int i = 0; i < SIZE; i++)
	{
		a[i] = i + 1;
	}
}

void host_init_b(int *b)
{
	for (int i = 0; i < N; i++)
	{
		b[i] = N - i;
	}
}
