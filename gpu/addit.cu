#include "slau.h"

void check_cuda_error(const char *msg)
{
    hipError_t err = hipGetLastError();

    if (hipSuccess != err)
	{
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
    }
}
