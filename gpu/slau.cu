#include "hip/hip_runtime.h"
#include "slau.h"

__global__ void search_det(double *a, double *det)
{
	__shared__ double divider;

	for (int k = 0; k < N - 1; k++)
	{
		if (threadIdx.x == k && blockIdx.x + k < N - 1)
		{
			if (blockIdx.x == 0 && a[N * threadIdx.x + threadIdx.x] == 0)
			{
				*det = 0;
				// return;
			}
			__syncthreads();
			// if (*det == 0)
			// 	return;
			if (*det != 0)
				divider = a[N * (blockIdx.x + threadIdx.x + 1) + threadIdx.x] / a[N * threadIdx.x + threadIdx.x];
		}
		__syncthreads();
		if (*det == 0)
			return;
		if (blockIdx.x + k < N - 1)
			a[N * (blockIdx.x + k + 1) + threadIdx.x] -= divider * a[N * k + threadIdx.x];
		__syncthreads();
	}
	if (blockIdx.x == 0 && threadIdx.x == 0)
	{
		for (int i = 0; i < N; i++)
		{
			*det *= a[N * i + i];
		}
		*det = __double2int_rn(*det);
	}
}

__global__ void search_minor_algaddit_matrix(double *a, int *minor_algaddit)
{
	__shared__ double sub_a[(N - 1) * (N - 1)];
	__shared__ double divider;
	__shared__ double det;

	if (!(threadIdx.x == blockIdx.x || threadIdx.y == blockIdx.y))
	{
		if (threadIdx.x > blockIdx.x && threadIdx.y > blockIdx.y)
			sub_a[N * threadIdx.y + threadIdx.x - (blockDim.y + 1) - (threadIdx.y - 1)] = a[N * threadIdx.y + threadIdx.x];
		else if (threadIdx.x < blockIdx.x && threadIdx.y < blockIdx.y)
			sub_a[N * threadIdx.y + threadIdx.x - threadIdx.y] = a[N * threadIdx.y + threadIdx.x];
		else if (threadIdx.x > blockIdx.x && threadIdx.y < blockIdx.y)
			sub_a[N * threadIdx.y + threadIdx.x - (threadIdx.y + 1)] = a[N * threadIdx.y + threadIdx.x];
		else
			sub_a[N * threadIdx.y + threadIdx.x - blockDim.y - (threadIdx.y - 1)] = a[N * threadIdx.y + threadIdx.x];
	}
	if (threadIdx.x == N - 1 || threadIdx.y != 0)
		return;
	__syncthreads();
	det = 1;
	for (int k = 0; k < (N - 1) - 1; k++)
	{
		if (threadIdx.x == k)
		{
			if (sub_a[(N - 1) * threadIdx.x + threadIdx.x] == 0)
			{
				printf("blockIdx.x = %d; blockIdx.y = %d\n", blockIdx.x, blockIdx.y);
				det = 0;
				// break;
			}
			// __syncthreads();
			// if (det == 0)
			// 	break;
			if (det != 0)
				divider = sub_a[(N - 1) * (threadIdx.x + 1) + threadIdx.x] / sub_a[(N - 1) * threadIdx.x + threadIdx.x];
		}
		__syncthreads();
		if (det == 0)
			break;
		sub_a[(N - 1) * (k + 1) + threadIdx.x] -= divider * sub_a[(N - 1) * k + threadIdx.x];
		__syncthreads();
	}
	if (threadIdx.x == 0)
	{
		if (det == 1)
		{
			for (int i = 0; i < (N - 1); i++)
			{
				det *= sub_a[(N - 1) * i + i];
			}
			det = __double2int_rn(det);
		}
	}
	__syncthreads();
	minor_algaddit[N * blockIdx.y + blockIdx.x] = det * pow(-1, blockIdx.x + blockIdx.y);
}

__global__ void transpose_matrix(int *a, double *at)
{
	int a_idx = N * (blockDim.y * blockIdx.y + threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;
	if (a_idx >= SIZE)
		return;
	int at_idx = N * (blockDim.x * blockIdx.x + threadIdx.x) + blockDim.y * blockIdx.y + threadIdx.y;
	if (at_idx >= SIZE)
		return;
	at[at_idx] = a[a_idx];
}

__global__ void get_inverse_matrix(double *a, double *det)
{
	int idx = N * (blockDim.y * blockIdx.y + threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= SIZE)
		return;
	int idx2 = N * (blockDim.x * blockIdx.x + threadIdx.x) + blockDim.y * blockIdx.y + threadIdx.y;
	if (idx2 >= SIZE)
		return;
	a[idx2] /= *det;
}

__global__ void mult_matrix_to_vector(double *a, int *b, double *x)
{
	int i0 = N * (blockDim.y * blockIdx.y + threadIdx.y);
	if (i0 >= SIZE)
		return;
	double sum = 0;
	for (int k = 0; k < N; k++)
		sum += a[i0 + k] * b[k];
	int idx = blockDim.y * blockIdx.y + threadIdx.y;
	x[idx] = sum;
}

int	main()
{
	// double  host_a[SIZE] = {2, 3, 4, 1}; // det = -10
	// double  host_a[SIZE] = {1, -2, 3, 4, 90, 6, -7, 8, 9}; // det = 2904
	// double host_a[SIZE] = {5, 3, 21, 7, 4, 47, 12, 18, 77, 45, 3, 1, -6, 90, 34, -82}; // det = 8765568
	// double host_a[SIZE] = {0, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16}; // det = 0
	// double host_a[SIZE] = {5, 3, 21, 7, 4, 47, 12, 18, 77, 45, 3, 1, -6, 90, 34, -82, -103, 71, 51, 21, 33, -367, 16, 2, 1}; // det = -1047253641
	double host_a[SIZE] = {5, 3, 21, 7, 4, 47, 0, 2, 3, 4, 12, 5, 6, 7, 8, 18, 9, 10, 11, 12, 77, 13, 14, 15, 16}; // det = -2332
	// int host_b[N] = {8, 6};
	// int host_b[N] = {8, 6, 17};
	// int host_b[N] = {8, 6, 17, 7};
	int host_b[N] = {8, 6, 17, 7, 9};

	// double *host_a;
	// int *host_b;
	double *host_x;
	int *host_minor_algaddit;
	double host_det;
	double *dev_a;
	int *dev_b;
	double *dev_x;
	double *dev_det;
	int *dev_minor_algaddit;
	int	int_size;
	int double_size;
	dim3 blocksPerGrid;
	dim3 threadsPerBlock;

	int_size = sizeof(int);
	double_size = sizeof(double);
	// host_a = (double *) malloc(double_size * SIZE);
	// host_b = (int *) malloc(int_size * N);
	// host_init_a(host_a);
	// host_init_b(host_b);
	host_print_matrix(host_a);
	host_print_vector(host_b);

	host_x = (double *) malloc(double_size * N);
	host_minor_algaddit = (int *) malloc(int_size * SIZE);
	host_det = 1;
	host_init_dim3(&blocksPerGrid, &threadsPerBlock);

	hipMalloc(&dev_a, double_size * SIZE);
	hipMalloc(&dev_b, int_size * N);
	hipMalloc(&dev_x, double_size * N);
	hipMalloc(&dev_det, double_size);
	hipMalloc(&dev_minor_algaddit, int_size * SIZE);

	hipMemcpy(dev_a, host_a, double_size * SIZE, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, host_b, int_size * N, hipMemcpyHostToDevice);
	hipMemcpy(dev_det, &host_det, double_size, hipMemcpyHostToDevice);

	search_det<<<N - 1, N>>>(dev_a, dev_det);
	hipMemcpy(&host_det, dev_det, double_size, hipMemcpyDeviceToHost);
	printf("Определитель матрицы = %f\n", host_det);
	if (host_det != 0)
	{
		hipMemcpy(dev_a, host_a, double_size * SIZE, hipMemcpyHostToDevice);
		search_minor_algaddit_matrix<<<dim3(N, N), dim3(N, N)>>>(dev_a, dev_minor_algaddit);
		hipMemcpy(host_minor_algaddit, dev_minor_algaddit, int_size * SIZE, hipMemcpyDeviceToHost);
		printf("Матрица алгебраических дополнений\n");
		host_print_matrix(host_minor_algaddit);
		transpose_matrix<<<blocksPerGrid, threadsPerBlock>>>(dev_minor_algaddit, dev_a);
		hipMemcpy(host_a, dev_a, double_size * SIZE, hipMemcpyDeviceToHost);
		printf("Транспонированная матрица\n");
		host_print_matrix(host_a);
		get_inverse_matrix<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_det);
		hipMemcpy(host_a, dev_a, double_size * SIZE, hipMemcpyDeviceToHost);
		printf("Обратная матрица\n");
		host_print_matrix(host_a);
		mult_matrix_to_vector<<<dim3(1, blocksPerGrid.y), dim3(1, threadsPerBlock.y)>>>(dev_a, dev_b, dev_x);
		hipMemcpy(host_x, dev_x, double_size * N, hipMemcpyDeviceToHost);
		printf("Ответ\n");
		host_print_vector(host_x);
	}
	else
		printf("Невозможно решить данную СЛАУ\n");

	// free(host_a);
	// free(host_b);
	free(host_x);
	free(host_minor_algaddit);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_x);
	hipFree(dev_det);
	hipFree(dev_minor_algaddit);

	host_check_cuda_error("");

	return 0;
}



// 2 * 2 - (1; 2)
// 3 * 3 - ()
// 4 * 4 - ()
// 2) 4 * 4 - ()
// 5 * 5 - ()
// 2) 5 * 5 - ()
