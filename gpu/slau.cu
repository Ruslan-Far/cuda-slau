#include "hip/hip_runtime.h"
#include "slau.h"

__global__ void	search_det(double *a, double *det)
{
	__shared__ double	divider;

	for (int k = 0; k < N - 1 && blockIdx.x < N - 1 - k; k++)
	{
		if (threadIdx.x == k)
		{
			if (blockIdx.x == 0 && a[N * threadIdx.x + threadIdx.x] == 0)
				*det = 0;
			__syncthreads();
			if (*det != 0)
				divider = a[N * (blockIdx.x + threadIdx.x + 1) + threadIdx.x] / a[N * threadIdx.x + threadIdx.x];
		}
		__syncthreads();
		if (*det == 0)
			return;
		a[N * (blockIdx.x + k + 1) + threadIdx.x] -= divider * a[N * k + threadIdx.x];
		__syncthreads();
	}
	if (blockIdx.x == 0 && threadIdx.x == 0)
	{
		for (int i = 0; i < N; i++)
			*det *= a[N * i + i];
		*det = __double2int_rn(*det);
	}
}

__global__ void	search_minor_algaddit_matrix(double *a, int *minor_algaddit)
{
	__shared__ double	sub_a[(N - 1) * (N - 1)];
	__shared__ double	divider[N - 1];
	__shared__ double	det;

	if (!(threadIdx.x == blockIdx.x || threadIdx.y == blockIdx.y))
	{
		if (threadIdx.x > blockIdx.x && threadIdx.y > blockIdx.y)
			sub_a[N * threadIdx.y + threadIdx.x - (blockDim.y + 1) - (threadIdx.y - 1)] = a[N * threadIdx.y + threadIdx.x];
		else if (threadIdx.x < blockIdx.x && threadIdx.y < blockIdx.y)
			sub_a[N * threadIdx.y + threadIdx.x - threadIdx.y] = a[N * threadIdx.y + threadIdx.x];
		else if (threadIdx.x > blockIdx.x && threadIdx.y < blockIdx.y)
			sub_a[N * threadIdx.y + threadIdx.x - (threadIdx.y + 1)] = a[N * threadIdx.y + threadIdx.x];
		else
			sub_a[N * threadIdx.y + threadIdx.x - blockDim.y - (threadIdx.y - 1)] = a[N * threadIdx.y + threadIdx.x];
	}
	if (threadIdx.x == N - 1 || threadIdx.y == N - 1)
		return;
	__syncthreads();
	det = 1;
	for (int k = 0; k < (N - 1) - 1 && threadIdx.y < N - 2 - k; k++)
	{
		if (threadIdx.x == k)
		{
			if (threadIdx.y == 0 && sub_a[(N - 1) * threadIdx.x + threadIdx.x] == 0)
				det = 0;
			__syncthreads();
			if (det != 0)
				divider[threadIdx.y] = sub_a[(N - 1) * (threadIdx.y + threadIdx.x + 1) + threadIdx.x] / sub_a[(N - 1) * threadIdx.x + threadIdx.x];
		}
		__syncthreads();
		if (det == 0)
			break;
		sub_a[(N - 1) * (threadIdx.y + k + 1) + threadIdx.x] -= divider[threadIdx.y] * sub_a[(N - 1) * k + threadIdx.x];
		__syncthreads();
	}
	if (threadIdx.x == 0 && threadIdx.y == 0)
	{
		if (det == 1)
		{
			for (int i = 0; i < (N - 1); i++)
				det *= sub_a[(N - 1) * i + i];
			det = __double2int_rn(det);
		}
	}
	if (threadIdx.x > 0 || threadIdx.y > 0)
		return;
	__syncthreads();
	minor_algaddit[N * blockIdx.y + blockIdx.x] = det * pow(-1, blockIdx.x + blockIdx.y);
}

__global__ void	transpose_matrix(int *a, double *at)
{
	int a_idx = N * (blockDim.y * blockIdx.y + threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;
	if (a_idx >= SIZE)
		return;
	int at_idx = N * (blockDim.x * blockIdx.x + threadIdx.x) + blockDim.y * blockIdx.y + threadIdx.y;
	if (at_idx >= SIZE)
		return;
	at[at_idx] = a[a_idx];
}

__global__ void	get_inverse_matrix(double *a, double *det)
{
	int idx = N * (blockDim.y * blockIdx.y + threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= SIZE)
		return;
	int idx2 = N * (blockDim.x * blockIdx.x + threadIdx.x) + blockDim.y * blockIdx.y + threadIdx.y;
	if (idx2 >= SIZE)
		return;
	a[idx2] /= *det;
}

__global__ void	mult_matrix_to_vector(double *a, int *b, double *x)
{
	int i0 = N * (blockDim.x * blockIdx.x + threadIdx.x);
	if (i0 >= SIZE)
		return;
	double sum = 0;
	for (int k = 0; k < N; k++)
		sum += a[i0 + k] * b[k];
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	x[idx] = sum;
}

int	main()
{
	double		*host_a;
	int			*host_b;
	double		*host_x;
	double		host_det;
	double		*dev_a;
	double		*dev_copy_a;
	int			*dev_b;
	double		*dev_x;
	double		*dev_det;
	int			*dev_minor_algaddit;
	int			int_size;
	int			double_size;
	dim3		blocksPerGrid;
	dim3		threadsPerBlock;
	hipEvent_t	start;
	hipEvent_t	stop;
	float		time;

	int_size = sizeof(int);
	double_size = sizeof(double);

	host_a = (double *) malloc(double_size * SIZE);
	host_b = (int *) malloc(int_size * N);
	host_init_a(host_a);
	host_init_b(host_b);
	printf("Матрица A\n");
	host_print_matrix(host_a);
	printf("Вектор B\n");
	host_print_vector(host_b);
	host_x = (double *) malloc(double_size * N);
	host_det = 1;
	host_init_dim3(&blocksPerGrid, &threadsPerBlock);

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMalloc(&dev_a, double_size * SIZE);
	hipMalloc(&dev_copy_a, double_size * SIZE);
	hipMalloc(&dev_b, int_size * N);
	hipMalloc(&dev_x, double_size * N);
	hipMalloc(&dev_det, double_size);
	hipMalloc(&dev_minor_algaddit, int_size * SIZE);

	hipMemcpy(dev_a, host_a, double_size * SIZE, hipMemcpyHostToDevice);
	hipMemcpy(dev_copy_a, host_a, double_size * SIZE, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, host_b, int_size * N, hipMemcpyHostToDevice);
	hipMemcpy(dev_det, &host_det, double_size, hipMemcpyHostToDevice);

	hipEventRecord(start, 0);
	search_det<<<N - 1, N>>>(dev_copy_a, dev_det);
	hipMemcpy(&host_det, dev_det, double_size, hipMemcpyDeviceToHost);
	if (host_det == 0)
	{
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
	}
	if (host_det != 0)
	{
		search_minor_algaddit_matrix<<<dim3(N, N), dim3(N, N)>>>(dev_a, dev_minor_algaddit);
		transpose_matrix<<<blocksPerGrid, threadsPerBlock>>>(dev_minor_algaddit, dev_a);
		get_inverse_matrix<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_det);
		mult_matrix_to_vector<<<blocksPerGrid.x, threadsPerBlock.x>>>(dev_a, dev_b, dev_x);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipMemcpy(host_x, dev_x, double_size * N, hipMemcpyDeviceToHost);
		printf("Ответ\n");
		host_print_vector(host_x);
	}
	else
		printf("Невозможно решить данную СЛАУ, так как определитель = 0\n");
	hipEventElapsedTime(&time, start, stop);
	printf("Время решения данной СЛАУ с %d неизвестными: %.2f мс\n", N, time);

	free(host_a);
	free(host_b);
	free(host_x);

	hipFree(dev_a);
	hipFree(dev_copy_a);
	hipFree(dev_b);
	hipFree(dev_x);
	hipFree(dev_det);
	hipFree(dev_minor_algaddit);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	host_check_cuda_error("");

	return 0;
}
