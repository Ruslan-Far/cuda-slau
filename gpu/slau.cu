#include "hip/hip_runtime.h"
#include "slau.h"

__constant__ int const_n;

__device__ int def_n(int n)
{
	if (n == 0)
		return N;
	return const_n;
}

__device__ void transform_matrix(double *a, int n)
{
	double divider;

	for (int k = 0; k < n - 1; k++)
	{
		for (int i = k; i < n - 1; i++)
		{
			if (a[n * k + k] == 0)
				break;
			divider = a[n * (i + 1) + k] / a[n * k + k];
			for (int j = 0; j < n; j++)
			{
				a[n * (i + 1) + k + j] -= divider * a[n * k + k + j];
			}
		}
	}
}

__device__ int get_det(double *a, int n)
{
	double det;

	n = def_n(n);
	// printf("n = %d\n", n);
	transform_matrix(a, n);
	// dev_print_matrix(a, n);
	det = 1;
	for (int i = 0; i < n; i++)
	{
		det *= a[n * i + i];
	}
	// printf("dev_det = %d\n", (int) round(det));
	return ((int) round(det));
}

__global__ void search_det(double *a, int *det)
{
	// *det = get_det(a, 0);



	__shared__ double divider;

	for (int k = 0; k < N - 1; k++)
	{
		if (threadIdx.x == k && blockIdx.x + k < N - 1)
		{
			if (blockIdx.x == 0 && a[N * threadIdx.x + threadIdx.x] == 0)
			{
				printf("Невозможно решить данную СЛАУ, так как определитель = 0\n");
				*det = 0;
				return;
			}
			__syncthreads();
			if (*det == 0)
				return;
			divider = a[N * (blockIdx.x + threadIdx.x + 1) + threadIdx.x] / a[N * threadIdx.x + threadIdx.x];
		}
		__syncthreads();
		if (blockIdx.x + k < N - 1)
			a[N * (blockIdx.x + k + 1) + threadIdx.x] -= divider * a[N * k + threadIdx.x];
		__syncthreads();
	}

	if (blockIdx.x == 0 && threadIdx.x == 0)
	{
		for (int i = 0; i < N; i++)
		{
			*det *= a[N * i + i];
		}
		*det = (int) (*det);
	}
	
}

__global__ void search_minor_algaddit_matrix(double *a, double *sub_a, int *minor_algaddit)
{
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			init_sub_a(a, sub_a, i, j);
			minor_algaddit[N * i + j] = get_det(sub_a, 1);
		}
	}
	printf("Матрица миноров\n");
	dev_print_matrix(minor_algaddit);
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			minor_algaddit[N * i + j] *= pow(-1, i + j);
		}
	}
}

__global__ void transpose_matrix(int *a, double *at)
{
	int a_idx = N * (blockDim.y * blockIdx.y + threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;
	if (a_idx >= SIZE)
	{
		// printf("a_idx = %d\n", a_idx);
		return;
	}
	// else printf("ELSE a_idx = %d; blockIdx.x = %d; threadIdx.x = %d; blockIdx.y = %d; threadIdx.y = %d\n", a_idx, blockIdx.x, threadIdx.x, blockIdx.y, threadIdx.y);
	int at_idx = N * (blockDim.x * blockIdx.x + threadIdx.x) + blockDim.y * blockIdx.y + threadIdx.y;
	if (at_idx >= SIZE)
	{
		// printf("at_idx = %d; blockIdx.x = %d; threadIdx.x = %d; blockIdx.y = %d; threadIdx.y = %d\n", at_idx, blockIdx.x, threadIdx.x, blockIdx.y, threadIdx.y);
		return;
	}
	// else printf("ELSE at_idx = %d; blockIdx.x = %d; threadIdx.x = %d; blockIdx.y = %d; threadIdx.y = %d\n", at_idx, blockIdx.x, threadIdx.x, blockIdx.y, threadIdx.y);
	at[at_idx] = a[a_idx];
}

__global__ void get_inverse_matrix(double *a, int *det)
{
	int idx = N * (blockDim.y * blockIdx.y + threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= SIZE)
		return;
	int idx2 = N * (blockDim.x * blockIdx.x + threadIdx.x) + blockDim.y * blockIdx.y + threadIdx.y;
	if (idx2 >= SIZE)
		return;
	// a[N * threadIdx.y + threadIdx.x] /= *det;
	// printf("idx = %d\n", idx);
	a[idx2] /= *det;
}

__global__ void mult_matrix_to_vector(double *a, int *b, double *x)
{
	int i0 = N * (blockDim.y * blockIdx.y + threadIdx.y);
	if (i0 >= SIZE)
	{
		// printf("blockDim.y = %d; blockIdx.y = %d; threadIdx.y = %d\n", blockDim.y, blockIdx.y, threadIdx.y);
		return;
	}
	double sum = 0;

	for (int k = 0; k < N; k++)
	{
		sum += a[i0 + k] * b[k];
	}
	int idx = blockDim.y * blockIdx.y + threadIdx.y;
	x[idx] = sum;
}

int	main()
{
	// double  host_a[SIZE] = {2, 3, 4, 1};
	// double  host_a[SIZE] = {1, -2, 3, 4, 90, 6, -7, 8, 9};
	double host_a[SIZE] = {5, 3, 21, 7, 4, 47, 12, 18, 77, 45, 3, 1, -6, 90, 34, -82};
	// double host_a[SIZE] = {5, 3, 21, 7, 4, 47, 12, 18, 77, 45, 3, 1, -6, 90, 34, -82, -103, 71, 51, 21, 33, -367, 16, 2, 1};
	// int host_b[N] = {8, 6};
	// int host_b[N] = {8, 6, 17};
	int host_b[N] = {8, 6, 17, 7};
	// double *host_a;
	// int *host_b;
	double *host_x;
	int *host_minor_algaddit;
	int host_det;
	double *dev_a;
	double *dev_sub_a;
	int *dev_b;
	double *dev_x;
	int *dev_det;
	int *dev_minor_algaddit;
	int host_n;
	int	int_size;
	int double_size;
	dim3 blocksPerGrid;
	dim3 threadsPerBlock;

	int_size = sizeof(int);
	double_size = sizeof(double);
	// host_a = (double *) malloc(double_size * SIZE);
	// host_b = (int *) malloc(int_size * N);
	// host_init_a(host_a);
	// host_init_b(host_b);
	host_print_matrix(host_a);
	host_print_vector(host_b);

	host_n = host_get_n();
	host_x = (double *) malloc(double_size * N);
	host_minor_algaddit = (int *) malloc(int_size * SIZE);
	host_det = 1;
	host_init_dim3(&blocksPerGrid, &threadsPerBlock);

	hipMalloc(&dev_a, double_size * SIZE);
	hipMalloc(&dev_sub_a, double_size * host_n);
	hipMalloc(&dev_b, int_size * N);
	hipMalloc(&dev_x, double_size * N);
	hipMalloc(&dev_det, int_size);
	hipMalloc(&dev_minor_algaddit, int_size * SIZE);

	hipMemcpyToSymbol(HIP_SYMBOL(const_n), &host_n, int_size, 0, hipMemcpyHostToDevice);
	hipMemcpy(dev_a, host_a, double_size * SIZE, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, host_b, int_size * N, hipMemcpyHostToDevice);
	hipMemcpy(dev_det, &host_det, int_size, hipMemcpyHostToDevice);

	search_det<<<3, 4>>>(dev_a, dev_det);
	// search_det<<<1, 1>>>(dev_a, dev_det);
	hipMemcpy(&host_det, dev_det, int_size, hipMemcpyDeviceToHost);
	printf("Определитель матрицы = %d\n", host_det);
	// if (host_det != 0)
	// {
	// 	hipMemcpy(dev_a, host_a, double_size * SIZE, hipMemcpyHostToDevice);
	// 	search_minor_algaddit_matrix<<<1, 1>>>(dev_a, dev_sub_a, dev_minor_algaddit);
	// 	hipMemcpy(host_minor_algaddit, dev_minor_algaddit, int_size * SIZE, hipMemcpyDeviceToHost);
	// 	printf("Матрица алгебраических дополнений\n");
	// 	host_print_matrix(host_minor_algaddit);
	// 	transpose_matrix<<<blocksPerGrid, threadsPerBlock>>>(dev_minor_algaddit, dev_a);
	// 	hipMemcpy(host_a, dev_a, double_size * SIZE, hipMemcpyDeviceToHost);
	// 	printf("Транспонированная матрица\n");
	// 	host_print_matrix(host_a);
	// 	get_inverse_matrix<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_det);
	// 	hipMemcpy(host_a, dev_a, double_size * SIZE, hipMemcpyDeviceToHost);
	// 	printf("Обратная матрица\n");
	// 	host_print_matrix(host_a);
	// 	mult_matrix_to_vector<<<dim3(1, blocksPerGrid.y), dim3(1, threadsPerBlock.y)>>>(dev_a, dev_b, dev_x);
	// 	hipMemcpy(host_x, dev_x, double_size * N, hipMemcpyDeviceToHost);
	// 	printf("Ответ\n");
	// 	host_print_vector(host_x);
	// }
	// else
	// 	printf("Невозможно решить данную СЛАУ\n");

	// free(host_a);
	// free(host_b);
	free(host_x);
	free(host_minor_algaddit);
	hipFree(dev_a);
	hipFree(dev_sub_a);
	hipFree(dev_b);
	hipFree(dev_x);
	hipFree(dev_det);
	hipFree(dev_minor_algaddit);

	host_check_cuda_error("");

	return 0;
}
