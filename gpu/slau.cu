#include "hip/hip_runtime.h"
#include "slau.h"

__constant__ int const_n;

__device__ int def_n(int n)
{
	if (n == 0)
		return N;
	return const_n;
}

__global__ void search_det(double *a, int *det)
{
	*det = get_det(a, 0);
}

__global__ void search_minor_algaddit_matrix(double *a, double *sub_a, int *minor_algaddit)
{
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			init_sub_a(a, sub_a, i, j);
			minor_algaddit[N * i + j] = get_det(sub_a, 1);
		}
	}
	printf("Матрица миноров\n");
	print_matrix(minor_algaddit);
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			minor_algaddit[N * i + j] *= pow(-1, i + j);
		}
	}
}

__global__ void transpose_matrix(int *a, double *at)
{
	int a_idx = N * (blockDim.y * blockIdx.y + threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;
	if (a_idx >= SIZE)
	{
		// printf("a_idx = %d\n", a_idx);
		return;
	}
	// else printf("ELSE a_idx = %d; blockIdx.x = %d; threadIdx.x = %d; blockIdx.y = %d; threadIdx.y = %d\n", a_idx, blockIdx.x, threadIdx.x, blockIdx.y, threadIdx.y);
	int at_idx = N * (blockDim.x * blockIdx.x + threadIdx.x) + blockDim.y * blockIdx.y + threadIdx.y;
	if (at_idx >= SIZE)
	{
		// printf("at_idx = %d; blockIdx.x = %d; threadIdx.x = %d; blockIdx.y = %d; threadIdx.y = %d\n", at_idx, blockIdx.x, threadIdx.x, blockIdx.y, threadIdx.y);
		return;
	}
	// else printf("ELSE at_idx = %d; blockIdx.x = %d; threadIdx.x = %d; blockIdx.y = %d; threadIdx.y = %d\n", at_idx, blockIdx.x, threadIdx.x, blockIdx.y, threadIdx.y);
	at[at_idx] = a[a_idx];
}

__global__ void get_inverse_matrix(double *a, int *det)
{
	int idx = N * (blockDim.y * blockIdx.y + threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= SIZE)
		return;
	int idx2 = N * (blockDim.x * blockIdx.x + threadIdx.x) + blockDim.y * blockIdx.y + threadIdx.y;
	if (idx2 >= SIZE)
		return;
	// a[N * threadIdx.y + threadIdx.x] /= *det;
	// printf("idx = %d\n", idx);
	a[idx2] /= *det;
}

__global__ void mult_matrix_to_vector(double *a, int *b, double *x)
{
	int i0 = N * (blockDim.y * blockIdx.y + threadIdx.y);
	if (i0 >= SIZE)
	{
		// printf("blockDim.y = %d; blockIdx.y = %d; threadIdx.y = %d\n", blockDim.y, blockIdx.y, threadIdx.y);
		return;
	}
	double sum = 0;

	for (int k = 0; k < N; k++)
	{
		sum += a[i0 + k] * b[k];
	}
	int idx = blockDim.y * blockIdx.y + threadIdx.y;
	x[idx] = sum;
}

void init_dim3(dim3 *blocksPerGrid, dim3 *threadsPerBlock)
{
	if (N <= BLOCK_N)
	{
		*blocksPerGrid = dim3(1);
		*threadsPerBlock = dim3(N, N);
	}
	else
	{
		if (N % BLOCK_N == 0)
			*blocksPerGrid = dim3(N / BLOCK_N, N / BLOCK_N);
		else
			*blocksPerGrid = dim3(N / BLOCK_N + 1, N / BLOCK_N + 1);
		*threadsPerBlock = dim3(BLOCK_N, BLOCK_N);
	}
}

int	main()
{
	// double  host_a[SIZE] = {2, 3, 4, 1};
	// double  host_a[SIZE] = {1, -2, 3, 4, 90, 6, -7, 8, 9};
	double host_a[SIZE] = {5, 3, 21, 7, 4, 47, 12, 18, 77, 45, 3, 1, -6, 90, 34, -82};
	// double host_a[SIZE] = {5, 3, 21, 7, 4, 47, 12, 18, 77, 45, 3, 1, -6, 90, 34, -82, -103, 71, 51, 21, 33, -367, 16, 2, 1};
	// int host_b[N] = {8, 6};
	// int host_b[N] = {8, 6, 17};
	int host_b[N] = {8, 6, 17, 7};
	double *host_x;
	int *host_minor_algaddit;
	int host_det;
	double *dev_a;
	double *dev_sub_a;
	int *dev_b;
	double *dev_x;
	int *dev_det;
	int *dev_minor_algaddit;
	int host_n;
	int	int_size;
	int double_size;
	dim3 blocksPerGrid;
	dim3 threadsPerBlock;

	int_size = sizeof(int);
	double_size = sizeof(double);
	host_n = get_n();
	host_x = (double *) malloc(double_size * N);
	host_minor_algaddit = (int *) malloc(int_size * SIZE);
	// blocksPerGrid = dim3(2, 2);
	// threadsPerBlock = dim3(BLOCK_N, BLOCK_N);
	init_dim3(&blocksPerGrid, &threadsPerBlock);

	hipMalloc(&dev_a, double_size * SIZE);
	hipMalloc(&dev_sub_a, double_size * host_n);
	hipMalloc(&dev_b, int_size * N);
	hipMalloc(&dev_x, double_size * N);
	hipMalloc(&dev_det, int_size);
	hipMalloc(&dev_minor_algaddit, int_size * SIZE);

	hipMemcpyToSymbol(HIP_SYMBOL(const_n), &host_n, int_size, 0, hipMemcpyHostToDevice);
	hipMemcpy(dev_a, host_a, double_size * SIZE, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, host_b, int_size * N, hipMemcpyHostToDevice);

	search_det<<<1, 1>>>(dev_a, dev_det);
	hipMemcpy(&host_det, dev_det, int_size, hipMemcpyDeviceToHost);
	printf("Определитель матрицы = %d\n", host_det);
	hipMemcpy(dev_a, host_a, double_size * SIZE, hipMemcpyHostToDevice);
	search_minor_algaddit_matrix<<<1, 1>>>(dev_a, dev_sub_a, dev_minor_algaddit);
	hipMemcpy(host_minor_algaddit, dev_minor_algaddit, int_size * SIZE, hipMemcpyDeviceToHost);
	printf("Матрица алгебраических дополнений\n");
	host_print_matrix(host_minor_algaddit);
	transpose_matrix<<<blocksPerGrid, threadsPerBlock>>>(dev_minor_algaddit, dev_a);
	hipMemcpy(host_a, dev_a, double_size * SIZE, hipMemcpyDeviceToHost);
	printf("Транспонированная матрица\n");
	host_print_matrix(host_a);
	// get_inverse_matrix<<<1, dim3(3, 3)>>>(dev_a, dev_det);
	get_inverse_matrix<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_det);
	hipMemcpy(host_a, dev_a, double_size * SIZE, hipMemcpyDeviceToHost);
	printf("Обратная матрица\n");
	host_print_matrix(host_a);
	// mult_matrix_to_vector<<<1, dim3(1, N)>>>(dev_a, dev_b, dev_x);
	mult_matrix_to_vector<<<dim3(1, blocksPerGrid.y), dim3(1, threadsPerBlock.y)>>>(dev_a, dev_b, dev_x);
	hipMemcpy(host_x, dev_x, double_size * N, hipMemcpyDeviceToHost);
	printf("Ответ\n");
	host_print_vector(host_x);

	free(host_x);
	free(host_minor_algaddit);
	hipFree(dev_a);
	hipFree(dev_sub_a);
	hipFree(dev_b);
	hipFree(dev_x);
	hipFree(dev_det);
	hipFree(dev_minor_algaddit);

	check_cuda_error("");

	return 0;
}
