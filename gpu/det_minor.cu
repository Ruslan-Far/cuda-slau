#include "hip/hip_runtime.h"
#include "slau.h"

int get_n()
{
	int k;

	k = 1;
	for (int i = 1; i < SIZE; i += k)
	{
		k += 2;
	}
	return sqrt(SIZE - k);
}

__device__ void transform_matrix(double *a, int n)
{
	double divider;

	for (int k = 0; k < n - 1; k++)
	{
		for (int i = k; i < n - 1; i++)
		{
			divider = a[n * (i + 1) + k] / a[n * k + k];
			for (int j = 0; j < n; j++)
			{
				a[n * (i + 1) + k + j] -= divider * a[n * k + k + j];
			}
		}
	}
}

__device__ int get_det(double *a, int n)
{
	double det;

	n = def_n(n);
	transform_matrix(a, n);
	// print_matrix(a, n);
	det = 1;
	for (int i = 0; i < n; i++)
	{
		det *= a[n * i + i];
	}
	// printf("det = %d\n", (int) round(det));
	return ((int) round(det));
}

__device__ void init_sub_a(double *a, double *sub_a, int r, int c)
{
	int idx_sub_a;

	idx_sub_a = 0;
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			if (i == r || j == c)
				continue;
			sub_a[idx_sub_a++] = a[N * i + j];
		}
	}
}
