#include "hip/hip_runtime.h"
#include "slau.h"

// __device__ void transform_matrix(double *a, int n)
// {
// 	double divider;

// 	for (int k = 0; k < n - 1; k++)
// 	{
// 		for (int i = k; i < n - 1; i++)
// 		{
// 			if (a[n * k + k] == 0)
// 				break;
// 			divider = a[n * (i + 1) + k] / a[n * k + k];
// 			for (int j = 0; j < n; j++)
// 			{
// 				a[n * (i + 1) + k + j] -= divider * a[n * k + k + j];
// 			}
// 		}
// 	}
// }

// __device__ int get_det(double *a, int n)
// {
// 	double det;

// 	n = def_n(n);
// 	// printf("n = %d\n", n);
// 	transform_matrix(a, n);
// 	// dev_print_matrix(a, n);
// 	det = 1;
// 	for (int i = 0; i < n; i++)
// 	{
// 		det *= a[n * i + i];
// 	}
// 	// printf("dev_det = %d\n", (int) round(det));
// 	return ((int) round(det));
// }

__device__ void init_sub_a(double *a, double *sub_a, int r, int c)
{
	int idx_sub_a;

	idx_sub_a = 0;
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			if (i == r || j == c)
				continue;
			sub_a[idx_sub_a++] = a[N * i + j];
		}
	}
}
