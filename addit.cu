#include "hip/hip_runtime.h"
#include "slau.h"

int get_n()
{
	int k;

	k = 1;
	for (int i = 1; i < SIZE; i += k)
	{
		k += 2;
	}
	return sqrt(SIZE - k);
}

void check_cuda_error(const char *msg)
{
    hipError_t err = hipGetLastError();

    if (hipSuccess != err)
	{
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
    }
}
