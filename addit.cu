#include "hip/hip_runtime.h"
#include "slau.h"

int get_n()
{
	int k;

	k = 1;
	for (int i = 1; i < SIZE; i += k)
	{
		k += 2;
	}
	return sqrt(SIZE - k);
}

void check_cuda_error(const char *msg)
{
    hipError_t err = hipGetLastError();

    if (hipSuccess != err)
	{
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
    }
}

void host_print_matrix(int *a)
{
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			printf("%d ", a[N * i + j]);
		}
		printf("\n");
	}
}
